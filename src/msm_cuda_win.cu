// visual studio compile notes, notice "YOUR" keyword 
// copy msm.fatbin to ~/.aleo/resources/cuda/msm.fatbin
// nvcc.exe -gencode=arch=compute_60,code=sm_60 -gencode=arch=compute_61,code=sm_61 -gencode=arch=compute_70,code=sm_70 -gencode=arch=compute_75,code=sm_75 -gencode=arch=compute_80,code=sm_80 -gencode=arch=compute_86,code=sm_86 --use-local-env -ccbin "YOUR\vs2022\IDE\VC\Tools\MSVC\14.33.31629\bin\HostX64\x64" -x cu   -I./ -I../../../Common -I./ -IYOUR\CUDA\/include -I../../../Common -IYOUR\CUDA\include     --keep-dir x64\Release  -maxrregcount=0  --machine 64 -fatbin -cudart static -Xcompiler "/wd 4819"  --threads 0 -DWIN32 -DWIN32 -D_MBCS -D_MBCS -o msm.fatbin "msm_cuda_win.cu"

#ifndef _ALEO_MSM_CUDA_H_
#define _ALEO_MSM_CUDA_H_

#include <stdio.h>

#ifdef __SIZE_TYPE__
typedef __SIZE_TYPE__ size_t;
#else
#include <stddef.h>
#endif

#include <stdint.h>

#include <string.h>
#include <hip/hip_runtime.h>

typedef unsigned long long limb_t;
# define LIMB_T_BITS    64

# define TO_LIMB_T(limb64)     limb64

#define NLIMBS(bits)   (bits/LIMB_T_BITS)
#define WINDOW_SIZE 128
// static const uint32_t BLST_WIDTH = 253;

typedef limb_t blst_scalar[NLIMBS(256)];
typedef limb_t blst_fr[NLIMBS(256)];
typedef limb_t blst_fp[NLIMBS(384)];
typedef limb_t vec768[NLIMBS(768)];

typedef struct { blst_fp X, Y; } blst_p1_affine;
typedef struct { blst_fp X, Y, Z; } blst_p1;
typedef struct { blst_fp X, Y, ZZ, ZZZ; } blst_p1_ext;

#define ONE_MONT_P TO_LIMB_T(0x02cdffffffffff68), \
                 TO_LIMB_T(0x51409f837fffffb1), \
                 TO_LIMB_T(0x9f7db3a98a7d3ff2), \
                 TO_LIMB_T(0x7b4e97b76e7c6305), \
                 TO_LIMB_T(0x4cf495bf803c84e8), \
                 TO_LIMB_T(0x008d6661e2fdf49a)

__device__ static const blst_fp BLS12_377_P = {
  TO_LIMB_T(0x8508c00000000001), TO_LIMB_T(0x170b5d4430000000),
  TO_LIMB_T(0x1ef3622fba094800), TO_LIMB_T(0x1a22d9f300f5138f),
  TO_LIMB_T(0xc63b05c06ca1493b), TO_LIMB_T(0x1ae3a4617c510ea)
};
__device__ static const blst_fp BLS12_377_ZERO{ 0 };
__device__ static const blst_fp BLS12_377_ONE{ ONE_MONT_P };
__device__ static const blst_fp BLS12_377_R2{
  0xb786686c9400cd22,
  0x329fcaab00431b1,
  0x22a5f11162d6b46d,
  0xbfdf7d03827dc3ac,
  0x837e92f041790bf9,
  0x6dfccb1e914b88,
};
__device__ static const limb_t BLS12_377_p0 = (limb_t)0x8508bfffffffffff;
__device__ extern const blst_p1 BLS12_377_ZERO_PROJECTIVE;
__device__ extern const blst_p1_affine BLS12_377_ZERO_AFFINE;
__device__ extern const blst_scalar BLS12_377_R;

__device__ static const blst_fp BIGINT_ONE = { 1, 0, 0, 0, 0, 0 };


__device__ const blst_p1 BLS12_377_ZERO_PROJECTIVE = {
  {0},
  {ONE_MONT_P},
  {0}
};

__device__ const blst_p1_affine BLS12_377_ZERO_AFFINE = {
  {0},
  {ONE_MONT_P}
};

__device__ const blst_scalar BLS12_377_R = {
  TO_LIMB_T(0x0a11800000000001), TO_LIMB_T(0x59aa76fed0000001),
  TO_LIMB_T(0x60b44d1e5c37b001), TO_LIMB_T(0x12ab655e9a2ca556)
};

__device__ static inline int is_blst_p1_zero(const blst_p1* p) {
    return p->Z[0] == 0 &&
        p->Z[1] == 0 &&
        p->Z[2] == 0 &&
        p->Z[3] == 0 &&
        p->Z[4] == 0 &&
        p->Z[5] == 0;
}

__device__ static inline int is_blst_fp_zero(const blst_fp p) {
    return p[0] == 0 &&
        p[1] == 0 &&
        p[2] == 0 &&
        p[3] == 0 &&
        p[4] == 0 &&
        p[5] == 0;
}

__device__ static inline int is_blst_fp_eq(const blst_fp p1, const blst_fp p2) {
    return p1[0] == p2[0] &&
        p1[1] == p2[1] &&
        p1[2] == p2[2] &&
        p1[3] == p2[3] &&
        p1[4] == p2[4] &&
        p1[5] == p2[5];
}

__device__ static inline int is_blst_p1_affine_zero(const blst_p1_affine* p) {
    return p->X[0] == 0 &&
        p->X[1] == 0 &&
        p->X[2] == 0 &&
        p->X[3] == 0 &&
        p->X[4] == 0 &&
        p->X[5] == 0;
}


// __device__ void mul_mont_384(blst_fp ret, const blst_fp a, const blst_fp b, const blst_fp p, limb_t p_inv);
// __device__ void sqr_mont_384(blst_fp ret, const blst_fp a, const blst_fp p, limb_t p_inv);
// __device__ void add_mod_384(blst_fp ret, const blst_fp a, const blst_fp b, const blst_fp p);
// __device__ void sub_mod_384(blst_fp ret, const blst_fp a, const blst_fp b, const blst_fp p);
// __device__ void sub_mod_384_unsafe(blst_fp ret, const blst_fp a, const blst_fp b);
// __device__ void add_mod_384_unsafe(blst_fp ret, const blst_fp a, const blst_fp b);
// __device__ void div_by_2_mod_384(blst_fp ret, const blst_fp a);
// __device__ void cneg_mod_384(blst_fp ret, const blst_fp a, bool flag, const blst_fp p);


__device__ static inline int is_gt_384(const blst_fp left, const blst_fp right) {
    for (int i = 5; i >= 0; --i) {
        if (left[i] < right[i]) {
            return 0;
        }
        else if (left[i] > right[i]) {
            return 1;
        }
    }
    return 0;
}

__device__ static inline int is_ge_384(const blst_fp left, const blst_fp right) {
    for (int i = 5; i >= 0; --i) {
        if (left[i] < right[i]) {
            return 0;
        }
        else if (left[i] > right[i]) {
            return 1;
        }
    }
    return 1;
}

__device__ static inline void sub_mod_384_unchecked(blst_fp ret, const blst_fp a, const blst_fp b) {
    asm(
        "sub.cc.u64 %0, %6, %12;\n\t"
        "subc.cc.u64 %1, %7, %13;\n\t"
        "subc.cc.u64 %2, %8, %14;\n\t"
        "subc.cc.u64 %3, %9, %15;\n\t"
        "subc.cc.u64 %4, %10, %16;\n\t"
        "subc.u64 %5, %11, %17;"
        : "=l"(ret[0]),
        "=l"(ret[1]),
        "=l"(ret[2]),
        "=l"(ret[3]),
        "=l"(ret[4]),
        "=l"(ret[5])
        : "l"(a[0]),
        "l"(a[1]),
        "l"(a[2]),
        "l"(a[3]),
        "l"(a[4]),
        "l"(a[5]),
        "l"(b[0]),
        "l"(b[1]),
        "l"(b[2]),
        "l"(b[3]),
        "l"(b[4]),
        "l"(b[5])
    );
    // return cf != 0?
}

__device__ static inline void reduce(blst_fp x, const blst_fp p) {
    if (is_ge_384(x, p)) {
        blst_fp x_sub;
        sub_mod_384_unchecked(x_sub, x, p);
        memcpy(x, x_sub, sizeof(blst_fp));
    }
}


// The Montgomery reduction here is based on Algorithm 14.32 in
// Handbook of Applied Cryptography
// <http://cacr.uwaterloo.ca/hac/about/chap14.pdf>.
__device__ static inline void mont_384(blst_fp ret, limb_t r[12], const blst_fp p, const limb_t p_inv) {
    // printf("c-t%i:0: %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, r[0], r[1], r[2], r[3], r[4], r[5], r[6], r[7], r[8], r[9], r[10], r[11]);
    limb_t k = r[0] * p_inv;

    limb_t cross_carry = 0;

    asm(
        "{\n\t"
        ".reg .u64 c;\n\t"
        ".reg .u64 t;\n\t"
        ".reg .u64 nc;\n\t"

        "mad.lo.cc.u64 c, %14, %8, %0;\n\t"
        "madc.hi.cc.u64 c, %14, %8, 0;\n\t"

        "addc.cc.u64 t, %1, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %1, %14, %9, t;\n\t"
        "madc.hi.cc.u64 c, %14, %9, nc;\n\t"

        "addc.cc.u64 t, %2, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %2, %14, %10, t;\n\t"
        "madc.hi.cc.u64 c, %14, %10, nc;\n\t"

        "addc.cc.u64 t, %3, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %3, %14, %11, t;\n\t"
        "madc.hi.cc.u64 c, %14, %11, nc;\n\t"

        "addc.cc.u64 t, %4, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %4, %14, %12, t;\n\t"
        "madc.hi.cc.u64 c, %14, %12, nc;\n\t"

        "addc.cc.u64 t, %5, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %5, %14, %13, t;\n\t"
        "madc.hi.cc.u64 c, %14, %13, nc;\n\t"

        "addc.cc.u64 %6, %6, c;\n\t"
        "addc.u64 %7, 0, 0;\n\t"
        "}"
        : "+l"(r[0]),
        "+l"(r[1]),
        "+l"(r[2]),
        "+l"(r[3]),
        "+l"(r[4]),
        "+l"(r[5]),
        "+l"(r[6]),
        "=l"(cross_carry)
        : "l"(p[0]),
        "l"(p[1]),
        "l"(p[2]),
        "l"(p[3]),
        "l"(p[4]),
        "l"(p[5]),
        "l"(k)
    );

    // printf("c-t%i:1: %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, r[0], r[1], r[2], r[3], r[4], r[5], r[6], r[7], r[8], r[9], r[10], r[11]);

    k = r[1] * p_inv;

    asm(
        "{\n\t"
        ".reg .u64 c;\n\t"
        ".reg .u64 t;\n\t"
        ".reg .u64 nc;\n\t"

        "mad.lo.cc.u64 c, %14, %8, %0;\n\t"
        "madc.hi.cc.u64 c, %14, %8, 0;\n\t"

        "addc.cc.u64 t, %1, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %1, %14, %9, t;\n\t"
        "madc.hi.cc.u64 c, %14, %9, nc;\n\t"

        "addc.cc.u64 t, %2, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %2, %14, %10, t;\n\t"
        "madc.hi.cc.u64 c, %14, %10, nc;\n\t"

        "addc.cc.u64 t, %3, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %3, %14, %11, t;\n\t"
        "madc.hi.cc.u64 c, %14, %11, nc;\n\t"

        "addc.cc.u64 t, %4, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %4, %14, %12, t;\n\t"
        "madc.hi.cc.u64 c, %14, %12, nc;\n\t"

        "addc.cc.u64 t, %5, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %5, %14, %13, t;\n\t"
        "madc.hi.cc.u64 c, %14, %13, nc;\n\t"

        "addc.cc.u64 c, c, %7;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "addc.cc.u64 %6, %6, c;\n\t"
        "addc.u64 %7, nc, 0;\n\t"
        "}"
        : "+l"(r[1]),
        "+l"(r[2]),
        "+l"(r[3]),
        "+l"(r[4]),
        "+l"(r[5]),
        "+l"(r[6]),
        "+l"(r[7]),
        "+l"(cross_carry)
        : "l"(p[0]),
        "l"(p[1]),
        "l"(p[2]),
        "l"(p[3]),
        "l"(p[4]),
        "l"(p[5]),
        "l"(k)
    );

    // printf("c-t%i:2: %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, r[0], r[1], r[2], r[3], r[4], r[5], r[6], r[7], r[8], r[9], r[10], r[11]);
    k = r[2] * p_inv;

    asm(
        "{\n\t"
        ".reg .u64 c;\n\t"
        ".reg .u64 t;\n\t"
        ".reg .u64 nc;\n\t"

        "mad.lo.cc.u64 c, %14, %8, %0;\n\t"
        "madc.hi.cc.u64 c, %14, %8, 0;\n\t"

        "addc.cc.u64 t, %1, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %1, %14, %9, t;\n\t"
        "madc.hi.cc.u64 c, %14, %9, nc;\n\t"

        "addc.cc.u64 t, %2, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %2, %14, %10, t;\n\t"
        "madc.hi.cc.u64 c, %14, %10, nc;\n\t"

        "addc.cc.u64 t, %3, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %3, %14, %11, t;\n\t"
        "madc.hi.cc.u64 c, %14, %11, nc;\n\t"

        "addc.cc.u64 t, %4, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %4, %14, %12, t;\n\t"
        "madc.hi.cc.u64 c, %14, %12, nc;\n\t"

        "addc.cc.u64 t, %5, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %5, %14, %13, t;\n\t"
        "madc.hi.cc.u64 c, %14, %13, nc;\n\t"

        "addc.cc.u64 c, c, %7;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "addc.cc.u64 %6, %6, c;\n\t"
        "addc.u64 %7, nc, 0;\n\t"
        "}"
        : "+l"(r[2]),
        "+l"(r[3]),
        "+l"(r[4]),
        "+l"(r[5]),
        "+l"(r[6]),
        "+l"(r[7]),
        "+l"(r[8]),
        "+l"(cross_carry)
        : "l"(p[0]),
        "l"(p[1]),
        "l"(p[2]),
        "l"(p[3]),
        "l"(p[4]),
        "l"(p[5]),
        "l"(k)
    );

    // printf("c-t%i:3: %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, r[0], r[1], r[2], r[3], r[4], r[5], r[6], r[7], r[8], r[9], r[10], r[11]);

    k = r[3] * p_inv;

    asm(
        "{\n\t"
        ".reg .u64 c;\n\t"
        ".reg .u64 t;\n\t"
        ".reg .u64 nc;\n\t"

        "mad.lo.cc.u64 c, %14, %8, %0;\n\t"
        "madc.hi.cc.u64 c, %14, %8, 0;\n\t"

        "addc.cc.u64 t, %1, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %1, %14, %9, t;\n\t"
        "madc.hi.cc.u64 c, %14, %9, nc;\n\t"

        "addc.cc.u64 t, %2, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %2, %14, %10, t;\n\t"
        "madc.hi.cc.u64 c, %14, %10, nc;\n\t"

        "addc.cc.u64 t, %3, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %3, %14, %11, t;\n\t"
        "madc.hi.cc.u64 c, %14, %11, nc;\n\t"

        "addc.cc.u64 t, %4, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %4, %14, %12, t;\n\t"
        "madc.hi.cc.u64 c, %14, %12, nc;\n\t"

        "addc.cc.u64 t, %5, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %5, %14, %13, t;\n\t"
        "madc.hi.cc.u64 c, %14, %13, nc;\n\t"

        "addc.cc.u64 c, c, %7;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "addc.cc.u64 %6, %6, c;\n\t"
        "addc.u64 %7, nc, 0;\n\t"
        "}"
        : "+l"(r[3]),
        "+l"(r[4]),
        "+l"(r[5]),
        "+l"(r[6]),
        "+l"(r[7]),
        "+l"(r[8]),
        "+l"(r[9]),
        "+l"(cross_carry)
        : "l"(p[0]),
        "l"(p[1]),
        "l"(p[2]),
        "l"(p[3]),
        "l"(p[4]),
        "l"(p[5]),
        "l"(k)
    );

    // printf("c-t%i:4: %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, r[0], r[1], r[2], r[3], r[4], r[5], r[6], r[7], r[8], r[9], r[10], r[11]);
    k = r[4] * p_inv;

    asm(
        "{\n\t"
        ".reg .u64 c;\n\t"
        ".reg .u64 t;\n\t"
        ".reg .u64 nc;\n\t"

        "mad.lo.cc.u64 c, %14, %8, %0;\n\t"
        "madc.hi.cc.u64 c, %14, %8, 0;\n\t"

        "addc.cc.u64 t, %1, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %1, %14, %9, t;\n\t"
        "madc.hi.cc.u64 c, %14, %9, nc;\n\t"

        "addc.cc.u64 t, %2, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %2, %14, %10, t;\n\t"
        "madc.hi.cc.u64 c, %14, %10, nc;\n\t"

        "addc.cc.u64 t, %3, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %3, %14, %11, t;\n\t"
        "madc.hi.cc.u64 c, %14, %11, nc;\n\t"

        "addc.cc.u64 t, %4, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %4, %14, %12, t;\n\t"
        "madc.hi.cc.u64 c, %14, %12, nc;\n\t"

        "addc.cc.u64 t, %5, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %5, %14, %13, t;\n\t"
        "madc.hi.cc.u64 c, %14, %13, nc;\n\t"

        "addc.cc.u64 c, c, %7;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "addc.cc.u64 %6, %6, c;\n\t"
        "addc.u64 %7, nc, 0;\n\t"
        "}"
        : "+l"(r[4]),
        "+l"(r[5]),
        "+l"(r[6]),
        "+l"(r[7]),
        "+l"(r[8]),
        "+l"(r[9]),
        "+l"(r[10]),
        "+l"(cross_carry)
        : "l"(p[0]),
        "l"(p[1]),
        "l"(p[2]),
        "l"(p[3]),
        "l"(p[4]),
        "l"(p[5]),
        "l"(k)
    );

    // printf("c-t%i:5: %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, r[0], r[1], r[2], r[3], r[4], r[5], r[6], r[7], r[8], r[9], r[10], r[11]);
    k = r[5] * p_inv;

    asm(
        "{\n\t"
        ".reg .u64 c;\n\t"
        ".reg .u64 t;\n\t"
        ".reg .u64 nc;\n\t"

        "mad.lo.cc.u64 c, %14, %8, %0;\n\t"
        "madc.hi.cc.u64 c, %14, %8, 0;\n\t"

        "addc.cc.u64 t, %1, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %1, %14, %9, t;\n\t"
        "madc.hi.cc.u64 c, %14, %9, nc;\n\t"

        "addc.cc.u64 t, %2, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %2, %14, %10, t;\n\t"
        "madc.hi.cc.u64 c, %14, %10, nc;\n\t"

        "addc.cc.u64 t, %3, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %3, %14, %11, t;\n\t"
        "madc.hi.cc.u64 c, %14, %11, nc;\n\t"

        "addc.cc.u64 t, %4, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %4, %14, %12, t;\n\t"
        "madc.hi.cc.u64 c, %14, %12, nc;\n\t"

        "addc.cc.u64 t, %5, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %5, %14, %13, t;\n\t"
        "madc.hi.cc.u64 c, %14, %13, nc;\n\t"

        "addc.cc.u64 c, c, %7;\n\t"
        // "addc.u64 nc, 0, 0;\n\t" if we dont want to clobber cross_carry we need this
        "add.u64 %6, %6, c;\n\t" // and this to be add.cc
        // "addc.u64 %7, nc, 0;\n\t" and this
        "}"
        : "+l"(r[5]),
        "+l"(r[6]),
        "+l"(r[7]),
        "+l"(r[8]),
        "+l"(r[9]),
        "+l"(r[10]),
        "+l"(r[11])
        : "l"(cross_carry),
        "l"(p[0]),
        "l"(p[1]),
        "l"(p[2]),
        "l"(p[3]),
        "l"(p[4]),
        "l"(p[5]),
        "l"(k)
    );

    // printf("c-t%i:6: %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, r[0], r[1], r[2], r[3], r[4], r[5], r[6], r[7], r[8], r[9], r[10], r[11]);

    memcpy(ret, r + 6, sizeof(limb_t) * 6);
    reduce(ret, p);
}

__device__ void mul_mont_384(blst_fp ret, const blst_fp a, const blst_fp b, const blst_fp p, limb_t p_inv) {
    limb_t r[12];

    asm(
        "{\n\t"
        ".reg .u64 c;\n\t"
        ".reg .u64 nc;\n\t"
        ".reg .u64 t;\n\t"

        "mad.lo.cc.u64 %0, %12, %18, 0;\n\t"
        "madc.hi.cc.u64 c, %12, %18, 0;\n\t"

        "madc.lo.cc.u64 %1, %12, %19, c;\n\t"
        "madc.hi.cc.u64 c, %12, %19, 0;\n\t"

        "madc.lo.cc.u64 %2, %12, %20, c;\n\t"
        "madc.hi.cc.u64 c, %12, %20, 0;\n\t"

        "madc.lo.cc.u64 %3, %12, %21, c;\n\t"
        "madc.hi.cc.u64 c, %12, %21, 0;\n\t"

        "madc.lo.cc.u64 %4, %12, %22, c;\n\t"
        "madc.hi.cc.u64 c, %12, %22, 0;\n\t"

        "madc.lo.cc.u64 %5, %12, %23, c;\n\t"
        "madc.hi.u64 %6, %12, %23, 0;\n\t"


        "mad.lo.cc.u64 %1, %13, %18, %1;\n\t"
        "madc.hi.cc.u64 c, %13, %18, 0;\n\t"

        "addc.cc.u64 t, %2, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %2, %13, %19, t;\n\t"
        "madc.hi.cc.u64 c, %13, %19, nc;\n\t"

        "addc.cc.u64 t, %3, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %3, %13, %20, t;\n\t"
        "madc.hi.cc.u64 c, %13, %20, nc;\n\t"

        "addc.cc.u64 t, %4, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %4, %13, %21, t;\n\t"
        "madc.hi.cc.u64 c, %13, %21, nc;\n\t"

        "addc.cc.u64 t, %5, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %5, %13, %22, t;\n\t"
        "madc.hi.cc.u64 c, %13, %22, nc;\n\t"

        "addc.cc.u64 t, %6, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %6, %13, %23, t;\n\t"
        "madc.hi.u64 %7, %13, %23, nc;\n\t"


        "mad.lo.cc.u64 %2, %14, %18, %2;\n\t"
        "madc.hi.cc.u64 c, %14, %18, 0;\n\t"

        "addc.cc.u64 t, %3, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %3, %14, %19, t;\n\t"
        "madc.hi.cc.u64 c, %14, %19, nc;\n\t"

        "addc.cc.u64 t, %4, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %4, %14, %20, t;\n\t"
        "madc.hi.cc.u64 c, %14, %20, nc;\n\t"

        "addc.cc.u64 t, %5, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %5, %14, %21, t;\n\t"
        "madc.hi.cc.u64 c, %14, %21, nc;\n\t"

        "addc.cc.u64 t, %6, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %6, %14, %22, t;\n\t"
        "madc.hi.cc.u64 c, %14, %22, nc;\n\t"

        "addc.cc.u64 t, %7, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %7, %14, %23, t;\n\t"
        "madc.hi.u64 %8, %14, %23, nc;\n\t"



        "mad.lo.cc.u64 %3, %15, %18, %3;\n\t"
        "madc.hi.cc.u64 c, %15, %18, 0;\n\t"

        "addc.cc.u64 t, %4, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %4, %15, %19, t;\n\t"
        "madc.hi.cc.u64 c, %15, %19, nc;\n\t"

        "addc.cc.u64 t, %5, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %5, %15, %20, t;\n\t"
        "madc.hi.cc.u64 c, %15, %20, nc;\n\t"

        "addc.cc.u64 t, %6, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %6, %15, %21, t;\n\t"
        "madc.hi.cc.u64 c, %15, %21, nc;\n\t"

        "addc.cc.u64 t, %7, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %7, %15, %22, t;\n\t"
        "madc.hi.cc.u64 c, %15, %22, nc;\n\t"

        "addc.cc.u64 t, %8, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %8, %15, %23, t;\n\t"
        "madc.hi.u64 %9, %15, %23, nc;\n\t"




        "mad.lo.cc.u64 %4, %16, %18, %4;\n\t"
        "madc.hi.cc.u64 c, %16, %18, 0;\n\t"

        "addc.cc.u64 t, %5, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %5, %16, %19, t;\n\t"
        "madc.hi.cc.u64 c, %16, %19, nc;\n\t"

        "addc.cc.u64 t, %6, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %6, %16, %20, t;\n\t"
        "madc.hi.cc.u64 c, %16, %20, nc;\n\t"

        "addc.cc.u64 t, %7, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %7, %16, %21, t;\n\t"
        "madc.hi.cc.u64 c, %16, %21, nc;\n\t"

        "addc.cc.u64 t, %8, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %8, %16, %22, t;\n\t"
        "madc.hi.cc.u64 c, %16, %22, nc;\n\t"

        "addc.cc.u64 t, %9, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %9, %16, %23, t;\n\t"
        "madc.hi.u64 %10, %16, %23, nc;\n\t"



        "mad.lo.cc.u64 %5, %17, %18, %5;\n\t"
        "madc.hi.cc.u64 c, %17, %18, 0;\n\t"

        "addc.cc.u64 t, %6, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %6, %17, %19, t;\n\t"
        "madc.hi.cc.u64 c, %17, %19, nc;\n\t"

        "addc.cc.u64 t, %7, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %7, %17, %20, t;\n\t"
        "madc.hi.cc.u64 c, %17, %20, nc;\n\t"

        "addc.cc.u64 t, %8, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %8, %17, %21, t;\n\t"
        "madc.hi.cc.u64 c, %17, %21, nc;\n\t"

        "addc.cc.u64 t, %9, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %9, %17, %22, t;\n\t"
        "madc.hi.cc.u64 c, %17, %22, nc;\n\t"

        "addc.cc.u64 t, %10, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %10, %17, %23, t;\n\t"
        "madc.hi.u64 %11, %17, %23, nc;\n\t"

        "}"
        : "+l"(r[0]),
        "+l"(r[1]),
        "+l"(r[2]),
        "+l"(r[3]),
        "+l"(r[4]),
        "+l"(r[5]),
        "+l"(r[6]),
        "+l"(r[7]),
        "+l"(r[8]),
        "+l"(r[9]),
        "+l"(r[10]),
        "+l"(r[11])
        : "l"(a[0]),
        "l"(a[1]),
        "l"(a[2]),
        "l"(a[3]),
        "l"(a[4]),
        "l"(a[5]),
        "l"(b[0]),
        "l"(b[1]),
        "l"(b[2]),
        "l"(b[3]),
        "l"(b[4]),
        "l"(b[5])
    );

    mont_384(ret, r, p, p_inv);
}

__device__ void sqr_mont_384(blst_fp ret, const blst_fp a, const blst_fp p, limb_t p_inv) {
    limb_t r[12];

    asm(
        "{\n\t"
        ".reg .u64 c;\n\t"
        ".reg .u64 nc;\n\t"
        ".reg .u64 t;\n\t"

        "mad.lo.cc.u64 %1, %12, %13, 0;\n\t"
        "madc.hi.cc.u64 c, %12, %13, 0;\n\t"

        "madc.lo.cc.u64 %2, %12, %14, c;\n\t"
        "madc.hi.cc.u64 c, %12, %14, 0;\n\t"

        "madc.lo.cc.u64 %3, %12, %15, c;\n\t"
        "madc.hi.cc.u64 c, %12, %15, 0;\n\t"

        "madc.lo.cc.u64 %4, %12, %16, c;\n\t"
        "madc.hi.cc.u64 c, %12, %16, 0;\n\t"

        "madc.lo.cc.u64 %5, %12, %17, c;\n\t"
        "madc.hi.u64 %6, %12, %17, 0;\n\t"

        "mad.lo.cc.u64 %3, %13, %14, %3;\n\t"
        "madc.hi.cc.u64 c, %13, %14, 0;\n\t"

        "addc.cc.u64 t, %4, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %4, %13, %15, t;\n\t"
        "madc.hi.cc.u64 c, %13, %15, nc;\n\t"

        "addc.cc.u64 t, %5, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %5, %13, %16, t;\n\t"
        "madc.hi.cc.u64 c, %13, %16, nc;\n\t"

        "addc.cc.u64 t, %6, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %6, %13, %17, t;\n\t"
        "madc.hi.u64 %7, %13, %17, nc;\n\t"



        "mad.lo.cc.u64 %5, %14, %15, %5;\n\t"
        "madc.hi.cc.u64 c, %14, %15, 0;\n\t"

        "addc.cc.u64 t, %6, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %6, %14, %16, t;\n\t"
        "madc.hi.cc.u64 c, %14, %16, nc;\n\t"

        "addc.cc.u64 t, %7, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %7, %14, %17, t;\n\t"
        "madc.hi.u64 %8, %14, %17, nc;\n\t"




        "mad.lo.cc.u64 %7, %15, %16, %7;\n\t"
        "madc.hi.cc.u64 c, %15, %16, 0;\n\t"

        "addc.cc.u64 t, %8, c;\n\t"
        "addc.u64 nc, 0, 0;\n\t"
        "mad.lo.cc.u64 %8, %15, %17, t;\n\t"
        "madc.hi.u64 %9, %15, %17, nc;\n\t"



        "mad.lo.cc.u64 %9, %16, %17, %9;\n\t"
        "madc.hi.u64 %10, %16, %17, 0;\n\t"

        "}"
        : "+l"(r[0]),
        "+l"(r[1]),
        "+l"(r[2]),
        "+l"(r[3]),
        "+l"(r[4]),
        "+l"(r[5]),
        "+l"(r[6]),
        "+l"(r[7]),
        "+l"(r[8]),
        "+l"(r[9]),
        "+l"(r[10]),
        "+l"(r[11])
        : "l"(a[0]),
        "l"(a[1]),
        "l"(a[2]),
        "l"(a[3]),
        "l"(a[4]),
        "l"(a[5])
    );

    // printf("c-t%i:0: X, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, X\n", threadIdx.x, r[1], r[2], r[3], r[4], r[5], r[6], r[7], r[8], r[9], r[10]);

    r[11] = r[10] >> 63;
    r[10] = (r[10] << 1) | (r[9] >> 63);
    r[9] = (r[9] << 1) | (r[8] >> 63);
    r[8] = (r[8] << 1) | (r[7] >> 63);
    r[7] = (r[7] << 1) | (r[6] >> 63);
    r[6] = (r[6] << 1) | (r[5] >> 63);
    r[5] = (r[5] << 1) | (r[4] >> 63);
    r[4] = (r[4] << 1) | (r[3] >> 63);
    r[3] = (r[3] << 1) | (r[2] >> 63);
    r[2] = (r[2] << 1) | (r[1] >> 63);
    r[1] = r[1] << 1;

    // printf("c-t%i:1: X, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, r[1], r[2], r[3], r[4], r[5], r[6], r[7], r[8], r[9], r[10], r[11]);

    asm(
        "{\n\t"

        "mad.lo.cc.u64 %0, %12, %12, 0;\n\t"
        "madc.hi.cc.u64 %1, %12, %12, %1;\n\t"

        "madc.lo.cc.u64 %2, %13, %13, %2;\n\t"
        "madc.hi.cc.u64 %3, %13, %13, %3;\n\t"

        "madc.lo.cc.u64 %4, %14, %14, %4;\n\t"
        "madc.hi.cc.u64 %5, %14, %14, %5;\n\t"

        "madc.lo.cc.u64 %6, %15, %15, %6;\n\t"
        "madc.hi.cc.u64 %7, %15, %15, %7;\n\t"

        "madc.lo.cc.u64 %8, %16, %16, %8;\n\t"
        "madc.hi.cc.u64 %9, %16, %16, %9;\n\t"

        "madc.lo.cc.u64 %10, %17, %17, %10;\n\t"
        "madc.hi.u64 %11, %17, %17, %11;\n\t"

        "}"
        : "+l"(r[0]),
        "+l"(r[1]),
        "+l"(r[2]),
        "+l"(r[3]),
        "+l"(r[4]),
        "+l"(r[5]),
        "+l"(r[6]),
        "+l"(r[7]),
        "+l"(r[8]),
        "+l"(r[9]),
        "+l"(r[10]),
        "+l"(r[11])
        : "l"(a[0]),
        "l"(a[1]),
        "l"(a[2]),
        "l"(a[3]),
        "l"(a[4]),
        "l"(a[5])
    );
    // printf("c-t%i:2: %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, r[0], r[1], r[2], r[3], r[4], r[5], r[6], r[7], r[8], r[9], r[10], r[11]);

    mont_384(ret, r, p, p_inv);
}


__device__ static inline void add_mod_384_unchecked(blst_fp ret, const blst_fp a, const blst_fp b) {
    asm(
        "add.cc.u64 %0, %6, %12;\n\t"
        "addc.cc.u64 %1, %7, %13;\n\t"
        "addc.cc.u64 %2, %8, %14;\n\t"
        "addc.cc.u64 %3, %9, %15;\n\t"
        "addc.cc.u64 %4, %10, %16;\n\t"
        "addc.u64 %5, %11, %17;"
        : "=l"(ret[0]),
        "=l"(ret[1]),
        "=l"(ret[2]),
        "=l"(ret[3]),
        "=l"(ret[4]),
        "=l"(ret[5])
        : "l"(a[0]),
        "l"(a[1]),
        "l"(a[2]),
        "l"(a[3]),
        "l"(a[4]),
        "l"(a[5]),
        "l"(b[0]),
        "l"(b[1]),
        "l"(b[2]),
        "l"(b[3]),
        "l"(b[4]),
        "l"(b[5])
    );
    // return cf != 0?
}

__device__ void add_mod_384(blst_fp ret, const blst_fp a, const blst_fp b, const blst_fp p) {
    add_mod_384_unchecked(ret, a, b);

    reduce(ret, p);
    // return cf != 0?
}

__device__ void sub_mod_384(blst_fp ret, const blst_fp a, const blst_fp b, const blst_fp p) {
    blst_fp added;
    memcpy(added, a, sizeof(blst_fp));
    // printf("pre-sub [%llu, %llu, %llu, %llu, %llu, %llu]\n", added[0], added[1], added[2], added[3], added[4], added[5]);
    if (is_gt_384(b, a)) {
        // printf("sub-preduce [%llu, %llu, %llu, %llu, %llu, %llu] > [%llu, %llu, %llu, %llu, %llu, %llu]\n", b[0], b[1], b[2], b[3], b[4], b[5], added[0], added[1], added[2], added[3], added[4], added[5]);
        add_mod_384_unchecked(added, added, p);
        // printf("sub-postduce [%llu, %llu, %llu, %llu, %llu, %llu]\n", added[0], added[1], added[2], added[3], added[4], added[5]);
    }
    else {
        // printf("sub-nonduce [%llu, %llu, %llu, %llu, %llu, %llu] <= [%llu, %llu, %llu, %llu, %llu, %llu]\n", b[0], b[1], b[2], b[3], b[4], b[5], added[0], added[1], added[2], added[3], added[4], added[5]);
    }
    sub_mod_384_unchecked(ret, added, b);
    // printf("post-sub [%llu, %llu, %llu, %llu, %llu, %llu]\n", ret[0], ret[1], ret[2], ret[3], ret[4], ret[5]);
    // return cf != 0?
}

__device__ void sub_mod_384_unsafe(blst_fp ret, const blst_fp a, const blst_fp b) {
    sub_mod_384_unchecked(ret, a, b);
    // return cf != 0?
}

__device__ void add_mod_384_unsafe(blst_fp ret, const blst_fp a, const blst_fp b) {
    add_mod_384_unchecked(ret, a, b);
    // return cf != 0?
}

__device__ static inline void _rshift_384(blst_fp ret, const blst_fp value) {
    ret[0] = (value[1] << 63) | (value[0] >> 1);
    ret[1] = (value[2] << 63) | (value[1] >> 1);
    ret[2] = (value[3] << 63) | (value[2] >> 1);
    ret[3] = (value[4] << 63) | (value[3] >> 1);
    ret[4] = (value[5] << 63) | (value[4] >> 1);
    ret[5] = value[5] >> 1;
}

__device__ void div_by_2_mod_384(blst_fp ret, const blst_fp a) {
    _rshift_384(ret, a);
}

__device__ void cneg_mod_384(blst_fp ret, const blst_fp a, bool flag, const blst_fp p) {
    // just let the compiler cmov
    if (flag) {
        sub_mod_384(ret, p, a, p);
    }
    else {
        memcpy(ret, a, 6 * sizeof(limb_t));
    }
}



__device__ static inline void blst_fp_add(blst_fp ret, const blst_fp a, const blst_fp b)
{
    add_mod_384(ret, a, b, BLS12_377_P);
}

__device__ static inline void blst_fp_add_unsafe(blst_fp ret, const blst_fp a, const blst_fp b)
{
    add_mod_384_unsafe(ret, a, b);
}

__device__ static inline void blst_fp_sub(blst_fp ret, const blst_fp a, const blst_fp b)
{
    sub_mod_384(ret, a, b, BLS12_377_P);
}

__device__ static inline void blst_fp_sub_unsafe(blst_fp ret, const blst_fp a, const blst_fp b)
{
    sub_mod_384_unsafe(ret, a, b);
}

__device__ static inline void blst_fp_cneg(blst_fp ret, const blst_fp a, bool flag)
{
    cneg_mod_384(ret, a, flag, BLS12_377_P);
}

__device__ static inline void blst_fp_mul(blst_fp ret, const blst_fp a, const blst_fp b)
{
    mul_mont_384(ret, a, b, BLS12_377_P, BLS12_377_p0);
}

__device__ static inline void blst_fp_sqr(blst_fp ret, const blst_fp a)
{
    sqr_mont_384(ret, a, BLS12_377_P, BLS12_377_p0);
}




__device__ void blst_fp_inverse(blst_fp out, const blst_fp in) {
    if (is_blst_fp_zero(in)) {
        // this is really bad
        *((int*)NULL);
    }
    // Guajardo Kumar Paar Pelzl
    // Efficient Software-Implementation of Finite Fields with Applications to
    // Cryptography
    // Algorithm 16 (BEA for Inversion in Fp)

    blst_fp u;
    memcpy(u, in, sizeof(blst_fp));
    blst_fp v;
    memcpy(v, BLS12_377_P, sizeof(blst_fp));
    blst_fp b;
    memcpy(b, BLS12_377_R2, sizeof(blst_fp));
    blst_fp c;
    memset(c, 0, sizeof(blst_fp));


    while (!is_blst_fp_eq(u, BIGINT_ONE) && !is_blst_fp_eq(v, BIGINT_ONE)) {
       // printf("c-t%i-inverse_round: u=%llu v=%llu b=%llu c=%llu\n", threadIdx.x, u[0], v[0], b[0], c[0]);
       while ((u[0] & 1) == 0) {
           // printf("c-t%i-inverse_round_u_start: u=%llu b=%llu\n", threadIdx.x, u[0], b[0]);
           div_by_2_mod_384(u, u);

           if ((b[0] & 1) != 0) {
               blst_fp_add_unsafe(b, b, BLS12_377_P);
           }
           div_by_2_mod_384(b, b);
           // printf("c-t%i-inverse_round_u_stop: u=%llu b=%llu\n", threadIdx.x, u[0], b[0]);
       }

       while ((v[0] & 1) == 0) {
           // printf("c-t%i-inverse_round_v_start: u=%llu b=%llu\n", threadIdx.x, v[0], c[0]);
           div_by_2_mod_384(v, v);

           if ((c[0] & 1) != 0) {
               blst_fp_add_unsafe(c, c, BLS12_377_P);
           }
           div_by_2_mod_384(c, c);
           // printf("c-t%i-inverse_round_v_stop: u=%llu b=%llu\n", threadIdx.x, v[0], c[0]);
       }

       if (is_gt_384(u, v)) {
           blst_fp_sub_unsafe(u, u, v);
           blst_fp_sub(b, b, c);
       }
       else {
           blst_fp_sub_unsafe(v, v, u);
           blst_fp_sub(c, c, b);
       }
    }
    if (is_blst_fp_eq(u, BIGINT_ONE)) {
        memcpy(out, b, sizeof(blst_fp));
    }
    else {
        memcpy(out, c, sizeof(blst_fp));
    }
}



__device__ void blst_p1_projective_into_affine(blst_p1_affine* out, const blst_p1* in) {
    if (is_blst_p1_zero(in)) {
        memset(out->X, 0, sizeof(blst_fp));
        memcpy(out->Y, BLS12_377_ONE, sizeof(blst_fp));
        //todo: set inf
    } else if (is_blst_fp_eq(in->Z, BLS12_377_ONE)) {
        memcpy(out->X, in->X, sizeof(blst_fp));
        memcpy(out->Y, in->Y, sizeof(blst_fp));
    } else {
        blst_fp z_inv;
        // printf("c-t%i:cinverse-in: %llu\n", threadIdx.x, in->Z[0]);
        blst_fp_inverse(z_inv, in->Z);
        // printf("c-t%i:cinverse-out: %llu\n", threadIdx.x, z_inv[0]);
        blst_fp z_inv_squared;
        blst_fp_sqr(z_inv_squared, z_inv);
        blst_fp_mul(out->X, in->X, z_inv_squared);
        blst_fp_mul(z_inv_squared, z_inv_squared, z_inv);
        blst_fp_mul(out->Y, in->Y, z_inv_squared);
    }
}

__device__ void blst_p1_double(blst_p1* out, const blst_p1* in) {
    if (is_blst_p1_zero(in)) {
        memcpy(out, in, sizeof(blst_p1));
    }

    // Z3 = 2*Y1*Z1
    blst_fp_mul(out->Z, in->Y, in->Z);
    blst_fp_add(out->Z, out->Z, out->Z);

    // A = X1^2
    blst_fp a;
    blst_fp_sqr(a, in->X);
    
    // B = Y1^2
    blst_fp b;
    blst_fp_sqr(b, in->Y);

    // C = B^2
    blst_fp c;
    blst_fp_sqr(c, b);

    // D = 2*((X1+B)^2-A-C)
    blst_fp d;
    blst_fp_add(d, in->X, b);
    blst_fp_sqr(d, d);
    blst_fp_sub(d, d, a);
    blst_fp_sub(d, d, c);
    blst_fp_add(d, d, d);

    // E = 3*A
    blst_fp e;
    blst_fp_add(e, a, a);
    blst_fp_add(e, e, a);

    // F = E^2
    blst_fp f;
    blst_fp_sqr(f, e);

    // X3 = F-2*D
    blst_fp_add(out->X, d, d);
    blst_fp_sub(out->X, f, out->X);

    // Y3 = E*(D-X3)-8*C
    blst_fp_sub(out->Y, d, out->X);
    blst_fp_mul(out->Y, out->Y, e);

    blst_fp c3;
    blst_fp_add(c3, c, c); // 2c
    blst_fp_add(c3, c3, c3); // 4c
    blst_fp_add(c3, c3, c3); // 8c
    blst_fp_sub(out->Y, out->Y, c3);
}

__device__ void blst_p1_double_affine(blst_p1* out, const blst_p1_affine* p) {
    /*
        dbl-2009-l from
        http://www.hyperelliptic.org/EFD/g1p/auto-shortw-jacobian-0.html#doubling-dbl-2009-l
    */

    // A = X1^2
    blst_fp A;
    blst_fp_sqr(A, p->X);

    // B = Y1^2
    blst_fp B;
    blst_fp_sqr(B, p->Y);

    // C = B^2
    blst_fp C;
    blst_fp_sqr(C, B);

    // D = 2 * ((X1 + B)^2 - A - C)
    blst_fp X1B;
    blst_fp_add(X1B, p->X, B);
    blst_fp_sqr(X1B, X1B);
    blst_fp_sub(X1B, X1B, A);
    blst_fp_sub(X1B, X1B, C);
    blst_fp D;
    blst_fp_add(D, X1B, X1B);

    // E = 3 * A
    blst_fp E;
    blst_fp_add(E, A, A);
    blst_fp_add(E, E, A);

    // F = E^2
    blst_fp F;
    blst_fp_sqr(F, E);

    // X3 = F - 2*D
    memcpy(out->X, F, sizeof(blst_fp));
    blst_fp_sub(out->X, out->X, D);
    blst_fp_sub(out->X, out->X, D);

    // Y3 = E*(D - X3) - 8*C
    blst_fp C8;
    blst_fp_add(C8, C, C);
    blst_fp_add(C8, C8, C8);
    blst_fp_add(C8, C8, C8);
    blst_fp_sub(D, D, out->X);
    blst_fp_mul(E, E, D);
    blst_fp_sub(out->Y, E, C8);

    // Z3 = 2*Y1
    blst_fp_add(out->Z, p->Y, p->Y);
}

__device__ void blst_p1_add_affine_to_projective(blst_p1 *out, const blst_p1 *p1, const blst_p1_affine *p2) {
    if (is_blst_p1_affine_zero(p2)) {
        memcpy(out, p1, sizeof(blst_p1));
        return;
    }

    if (is_blst_p1_zero(p1)) {
        memcpy(out->X, p2->X, sizeof(blst_fp));
        memcpy(out->Y, p2->Y, sizeof(blst_fp));
        memcpy(out->Z, BLS12_377_ONE, sizeof(blst_fp));
        return;
    }
  
    // http://www.hyperelliptic.org/EFD/g1p/auto-shortw-jacobian-0.html#addition-madd-2007-bl
    // Works for all curves.

    // printf("c-t%llu:add:0 %llu,%llu,%llu -> %llu,%llu\n", threadIdx.x, p1->X[0], p1->Y[0], p1->Z[0], p2->X[0], p2->Y[0]);

    // Z1Z1 = Z1^2
    blst_fp z1z1;
    blst_fp_sqr(z1z1, p1->Z);

    // printf("c-t%llu:add:1 %llu\n", threadIdx.x, z1z1[0]);

    // U2 = X2*Z1Z1
    blst_fp u2;
    blst_fp_mul(u2, p2->X, z1z1);

    // printf("c-t%llu:add:2 %llu\n", threadIdx.x, u2[0]);

    // S2 = Y2*Z1*Z1Z1
    blst_fp s2;
    blst_fp_mul(s2, p2->Y, p1->Z);
    blst_fp_mul(s2, s2, z1z1);

    if (is_blst_fp_eq(p1->X, u2) && is_blst_fp_eq(p1->Y, s2)) {
        blst_p1_double(out, p1);
        return;
    }

    // printf("c-t%llu:add:3 %llu\n", threadIdx.x, s2[0]);

    // printf("c-t%llu:add:pre-4 %llu - %llu\n", threadIdx.x, u2[0], p1->X[0]);
    // H = U2-X1
    blst_fp h;
    blst_fp_sub(h, u2, p1->X);

    // printf("c-t%llu:add:4 %llu\n", threadIdx.x, h[0]);

    // HH = H^2
    blst_fp hh;
    blst_fp_sqr(hh, h);
    // printf("c-t%llu:add:5 %llu\n", threadIdx.x, hh[0]);

    // I = 4*HH
    blst_fp i;
    memcpy(i, hh, sizeof(blst_fp));
    blst_fp_add(i, i, i);
    blst_fp_add(i, i, i);
    // printf("c-t%llu:add:6 %llu\n", threadIdx.x, i[0]);

    // J = H*I
    blst_fp j;
    blst_fp_mul(j, h, i);
    // printf("c-t%llu:add:7 %llu\n", threadIdx.x, j[0]);

    // r = 2*(S2-Y1)
    blst_fp r;
    blst_fp_sub(r, s2, p1->Y);
    blst_fp_add(r, r, r);
    // printf("c-t%llu:add:8 %llu\n", threadIdx.x, r[0]);

    // V = X1*I
    blst_fp v;
    blst_fp_mul(v, p1->X, i);
    // printf("c-t%llu:add:9 %llu\n", threadIdx.x, v[0]);

    // X3 = r^2 - J - 2*V
    blst_fp_sqr(out->X, r);
    // printf("c-t%llu:add:1X %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, out->X[0], out->X[1], out->X[2], out->X[3], out->X[4], out->X[5]);
    blst_fp_sub(out->X, out->X, j);
    // printf("c-t%llu:add:2X %llu, %llu, %llu, %llu, %llu, %llu -- %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, out->X[0], out->X[1], out->X[2], out->X[3], out->X[4], out->X[5], j[0], j[1], j[2], j[3], j[4], j[5]);
    blst_fp_sub(out->X, out->X, v);
    // printf("c-t%llu:add:3X %llu\n", threadIdx.x, out->X[0]);
    blst_fp_sub(out->X, out->X, v);
    // printf("c-t%llu:add:4X %llu\n", threadIdx.x, out->X[0]);

    // Y3 = r*(V-X3)-2*Y1*J
    blst_fp_mul(j, p1->Y, j);
    blst_fp_add(j, j, j);
    blst_fp_sub(out->Y, v, out->X);
    blst_fp_mul(out->Y, out->Y, r);
    blst_fp_sub(out->Y, out->Y, j);
    // printf("c-t%llu:add:Y %llu\n", threadIdx.x, out->Y[0]);

    // Z3 = (Z1+H)^2-Z1Z1-HH
    blst_fp_add(out->Z, p1->Z, h);
    blst_fp_sqr(out->Z, out->Z);
    blst_fp_sub(out->Z, out->Z, z1z1);
    blst_fp_sub(out->Z, out->Z, hh);
    // printf("c-t%llu:add:Z %llu\n", threadIdx.x, out->Z[0]);
}


__device__ void blst_p1_add_affines_into_projective(blst_p1* out, const blst_p1_affine* p1, const blst_p1_affine* p2) {
    /*
        mmadd-2007-bl from
        http://www.hyperelliptic.org/EFD/g1p/auto-shortw-jacobian-0.html#addition-mmadd-2007-bl
    */

    // The points can't be 0.
    if (is_blst_p1_affine_zero(p2)) {
        memcpy(out->X, p1->X, sizeof(blst_fp));
        memcpy(out->Y, p1->Y, sizeof(blst_fp));

        if (is_blst_p1_affine_zero(p1)) {
            memcpy(out->Z, BLS12_377_ZERO, sizeof(blst_fp));
        } else {
            memcpy(out->Z, BLS12_377_ONE, sizeof(blst_fp));
        }

        return;
    } else if (is_blst_p1_affine_zero(p1)) {
        memcpy(out->X, p2->X, sizeof(blst_fp));
        memcpy(out->Y, p2->Y, sizeof(blst_fp));

        if (is_blst_p1_affine_zero(p2)) {
            memcpy(out->Z, BLS12_377_ZERO, sizeof(blst_fp));
        } else {
            memcpy(out->Z, BLS12_377_ONE, sizeof(blst_fp));
        }

        return;
    }

    // mmadd-2007-bl does not support equal values for p1 and p2.
    // If `p1` and `p2` are equal, use the doubling algorithm.
    if(is_blst_fp_eq(p1->X, p2->X) && is_blst_fp_eq(p1->Y, p2->Y)) {
        blst_p1_double_affine(out, p1);
        return;
    }

    // H = X2-X1
    blst_fp h;
    blst_fp_sub(h, p2->X, p1->X);

    // HH = H^2
    // I = 4*HH
    blst_fp i;
    memcpy(i, h, sizeof(blst_fp));
    blst_fp_add(i, i, i);
    blst_fp_sqr(i, i);

    // J = H*I
    blst_fp j;
    blst_fp_mul(j, h, i);

    // r = 2*(Y2-Y1)
    blst_fp r;
    blst_fp_sub(r, p2->Y, p1->Y);
    blst_fp_add(r, r, r);

    // V = X1*I
    blst_fp v;
    blst_fp_mul(v, p1->X, i);

    // X3 = r^2-J-2*V
    blst_fp_sqr(out->X, r);
    blst_fp_sub(out->X, out->X, j);
    blst_fp_sub(out->X, out->X, v);
    blst_fp_sub(out->X, out->X, v);

    // Y3 = r*(V-X3)-2*Y1*J
    blst_fp_sub(out->Y, v, out->X);
    blst_fp_mul(out->Y, out->Y, r);

    blst_fp y1j;
    blst_fp_mul(y1j, p1->Y, j);
    blst_fp_sub(out->Y, out->Y, y1j);
    blst_fp_sub(out->Y, out->Y, y1j);

    // Z3 = 2*H
    blst_fp_add(out->Z, h, h);
}

__device__ void blst_p1_add_projective_to_projective(blst_p1 *out, const blst_p1 *p1, const blst_p1 *p2) {
    if (is_blst_p1_zero(p2)) {
        memcpy(out, p1, sizeof(blst_p1));
        return;
    }

    if (is_blst_p1_zero(p1)) {
        memcpy(out, p2, sizeof(blst_p1));
        return;
    }

    int p1_is_affine = is_blst_fp_eq(p1->Z, BLS12_377_ONE);
    int p2_is_affine = is_blst_fp_eq(p2->Z, BLS12_377_ONE);
    // //todo: confirm generated ptx here is *okay* for warp divergence
    if (p1_is_affine && p2_is_affine) {
        blst_p1_affine p1_affine;
        memcpy(&p1_affine.X, &p1->X, sizeof(blst_fp));
        memcpy(&p1_affine.Y, &p1->Y, sizeof(blst_fp));
        blst_p1_affine p2_affine;
        memcpy(&p2_affine.X, &p2->X, sizeof(blst_fp));
        memcpy(&p2_affine.Y, &p2->Y, sizeof(blst_fp));
        blst_p1_add_affines_into_projective(out, &p1_affine, &p2_affine);
        return;
    } if (p1_is_affine) {
        blst_p1_affine p1_affine;
        memcpy(&p1_affine.X, &p1->X, sizeof(blst_fp));
        memcpy(&p1_affine.Y, &p1->Y, sizeof(blst_fp));
        blst_p1_add_affine_to_projective(out, p2, &p1_affine);
        return;
    } else if (p2_is_affine) {
        blst_p1_affine p2_affine;
        memcpy(&p2_affine.X, &p2->X, sizeof(blst_fp));
        memcpy(&p2_affine.Y, &p2->Y, sizeof(blst_fp));
        blst_p1_add_affine_to_projective(out, p1, &p2_affine);
        return;
    }
  
    // http://www.hyperelliptic.org/EFD/g1p/auto-shortw-jacobian-0.html#addition-madd-2007-bl
    // Works for all curves.

    // printf("c-t%llu:add:0 %llu,%llu,%llu -> %llu,%llu\n", threadIdx.x, p1->X[0], p1->Y[0], p1->Z[0], p2->X[0], p2->Y[0]);

    // Z1Z1 = Z1^2
    blst_fp z1z1;
    blst_fp_sqr(z1z1, p1->Z);

    // Z2Z2 = Z2^2
    blst_fp z2z2;
    blst_fp_sqr(z2z2, p2->Z);

    // U1 = X1*Z2Z2
    blst_fp u1;
    blst_fp_mul(u1, p1->X, z2z2);

    // U2 = X2*Z1Z1
    blst_fp u2;
    blst_fp_mul(u2, p2->X, z1z1);

    // S1 = Y1*Z2*Z2Z2
    blst_fp s1;
    blst_fp_mul(s1, p1->Y, p2->Z);
    blst_fp_mul(s1, s1, z2z2);

    // S2 = Y2*Z1*Z1Z1
    blst_fp s2;
    blst_fp_mul(s2, p2->Y, p1->Z);
    blst_fp_mul(s2, s2, z1z1);

    // H = U2-U1
    blst_fp h;
    blst_fp_sub(h, u2, u1);

    // printf("c-t%llu:add:4 %llu\n", threadIdx.x, h[0]);

    // HH = H^2
    blst_fp hh;
    blst_fp_sqr(hh, h);
    // printf("c-t%llu:add:5 %llu\n", threadIdx.x, hh[0]);

    // I = 4*HH
    blst_fp i;
    memcpy(i, hh, sizeof(blst_fp));
    blst_fp_add(i, i, i);
    blst_fp_add(i, i, i);
    // printf("c-t%llu:add:6 %llu\n", threadIdx.x, i[0]);

    // J = H*I
    blst_fp j;
    blst_fp_mul(j, h, i);
    // printf("c-t%llu:add:7 %llu\n", threadIdx.x, j[0]);

    // r = 2*(S2-S1)
    blst_fp r;
    blst_fp_sub(r, s2, s1);
    blst_fp_add(r, r, r);
    // printf("c-t%llu:add:8 %llu\n", threadIdx.x, r[0]);

    // V = U1*I
    blst_fp v;
    blst_fp_mul(v, u1, i);
    // printf("c-t%llu:add:9 %llu\n", threadIdx.x, v[0]);

    // X3 = r^2 - J - 2*V
    blst_fp_sqr(out->X, r);
    // printf("c-t%llu:add:1X %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, out->X[0], out->X[1], out->X[2], out->X[3], out->X[4], out->X[5]);
    blst_fp_sub(out->X, out->X, j);
    // printf("c-t%llu:add:2X %llu, %llu, %llu, %llu, %llu, %llu -- %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, out->X[0], out->X[1], out->X[2], out->X[3], out->X[4], out->X[5], j[0], j[1], j[2], j[3], j[4], j[5]);
    blst_fp_sub(out->X, out->X, v);
    // printf("c-t%llu:add:3X %llu\n", threadIdx.x, out->X[0]);
    blst_fp_sub(out->X, out->X, v);
    // printf("c-t%llu:add:4X %llu\n", threadIdx.x, out->X[0]);

    // Y3 = r*(V-X3)-2*S1*J
    blst_fp_mul(j, s1, j);
    blst_fp_add(j, j, j);
    blst_fp_sub(out->Y, v, out->X);
    blst_fp_mul(out->Y, out->Y, r);
    blst_fp_sub(out->Y, out->Y, j);
    // printf("c-t%llu:add:Y %llu\n", threadIdx.x, out->Y[0]);

    // Z3 = ((Z1+Z2)^2-Z1Z1-Z2Z2)*H
    blst_fp_add(out->Z, p1->Z, p2->Z);
    blst_fp_sqr(out->Z, out->Z);
    blst_fp_sub(out->Z, out->Z, z1z1);
    blst_fp_sub(out->Z, out->Z, z2z2);
    blst_fp_mul(out->Z, out->Z, h);
    // printf("c-t%llu:add:Z %llu\n", threadIdx.x, out->Z[0]);
}


__device__ void blst_p1_add_affine_to_affine(blst_p1_affine* out, const blst_p1_affine* p1, const blst_p1_affine* p2) {
    /*
        http://www.hyperelliptic.org/EFD/g1p/auto-shortw.html
        x3 = (y2-y1)2/(x2-x1)2-x1-x2
        y3 = (2*x1+x2)*(y2-y1)/(x2-x1)-(y2-y1)3/(x2-x1)3-y1
    */
    blst_fp y_diff;
    blst_fp_sub(y_diff, p2->Y, p1->Y);

    blst_fp y_diff2;
    blst_fp_sqr(y_diff2, y_diff);

    blst_fp x_diff_inv;
    blst_fp_sub(x_diff_inv, p2->X, p1->X);
    blst_fp_inverse(x_diff_inv, x_diff_inv);
    
    blst_fp x_diff_inv2;
    blst_fp_sqr(x_diff_inv2, x_diff_inv);

    blst_fp sum_x;
    blst_fp_add(sum_x, p1->X, p2->X);

    blst_fp_mul(out->X, y_diff2, x_diff_inv2);
    blst_fp_sub(out->X, out->X, sum_x);

    blst_fp_mul(out->Y, y_diff, x_diff_inv);
    blst_fp_mul(out->Y, out->Y, sum_x);
    blst_fp_add(out->Y, out->Y, out->Y);

    blst_fp y_diff3;
    blst_fp_mul(y_diff3, y_diff2, y_diff);

    blst_fp x_diff_inv3;
    blst_fp_mul(x_diff_inv3, x_diff_inv2, x_diff_inv);

    blst_fp j;
    blst_fp_mul(j, y_diff3, x_diff_inv3);
    blst_fp_sub(out->Y, out->Y, j);

    blst_fp_sub(out->Y, out->Y, p1->Y);
}

extern "C" __global__ void msm6_pixel(blst_p1 * bucket_lists, const blst_p1_affine * bases_in, const blst_scalar * scalars, const uint32_t * window_lengths, const uint32_t window_count) {
   limb_t index = threadIdx.x / 64;
   size_t shift = threadIdx.x - (index * 64);
   limb_t mask = (limb_t)1 << (limb_t)shift;

   blst_p1 bucket;
   memcpy(&bucket, &BLS12_377_ZERO_PROJECTIVE, sizeof(blst_p1));

   uint32_t window_start = WINDOW_SIZE * blockIdx.x;
   uint32_t window_end = window_start + window_lengths[blockIdx.x];

   uint32_t activated_bases[WINDOW_SIZE];
   uint32_t activated_base_index = 0;

   // we delay the actual additions to a second loop because it reduces warp divergence (20% practical gain)
   for (uint32_t i = window_start; i < window_end; ++i) {
       limb_t bit = (scalars[i][index] & mask);
       if (bit == 0) {
           continue;
       }
       activated_bases[activated_base_index++] = i;
   }
   uint32_t i = 0;
   for (; i < (activated_base_index / 2 * 2); i += 2) {
       blst_p1 intermediate;
       blst_p1_add_affines_into_projective(&intermediate, &bases_in[activated_bases[i]], &bases_in[activated_bases[i + 1]]);
       blst_p1_add_projective_to_projective(&bucket, &bucket, &intermediate);
   }
   for (; i < activated_base_index; ++i) {
       blst_p1_add_affine_to_projective(&bucket, &bucket, &(bases_in[activated_bases[i]]));
   }

   memcpy(&bucket_lists[threadIdx.x * window_count + blockIdx.x], &bucket, sizeof(blst_p1));
}

extern "C" __global__ void msm6_collapse_rows(blst_p1 * target, const blst_p1 * bucket_lists, const uint32_t window_count) {
    blst_p1 temp_target;
    uint32_t base = threadIdx.x * window_count;
    uint32_t term = base + window_count;
    memcpy(&temp_target, &bucket_lists[base], sizeof(blst_p1));

    for (uint32_t i = base + 1; i < term; ++i) {
       blst_p1_add_projective_to_projective(&temp_target, &temp_target, &bucket_lists[i]);
    }

    memcpy(&target[threadIdx.x], &temp_target, sizeof(blst_p1));
}

#endif  // #ifndef _ALEO_MSM_CUDA_H_
